//Parallelization - N blocks 1 thread/block

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <iostream>


#define N 32

__global__ void add(int *a, int *b, int *c)
{
        //block id
        c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random (int *tab, int wym )
{
        int i;
        for(i=0;i<wym;i++)
                tab[i]=rand()%101;
}


int main(void) {
        int *a, *b, *c; // host copies of a, b, c
        int *d_a, *d_b, *d_c; // device copies of a, b, c
        int size = N * sizeof(int);
        int i;
        srand(time(NULL));
        // Allocate space for device copies of a, b, c
        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);
        // Alloc space for host copies of a, b, c and setup input values
        a = (int *)malloc(size); random(a, N);
        b = (int *)malloc(size); random(b, N);
        c = (int *)malloc(size);
        // Copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
        // Run kernel - N blocks - 1 thread
        add<<<N,1>>>(d_a, d_b, d_c);
        // Copy result back to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

        for(i=0;i<N;i++)
        {
                printf("a[%d](%d) + b[%d](%d) = c[%d](%d)\n",i,a[i],i,b[i],i,c[i]);
        }
        // Cleanup
        free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        return 0;
}