#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <iostream>


//GPU function (kernel)

__global__ void add(int *a, int *b, int *c) 
{
	*c = *a + *b;
}

int main(void) {
	int a, b, c; // host side tables
	int *d_a, *d_b, *d_c; // device side tables
	int size = sizeof(int);
	// CUDA device memory allocation
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Example values
	a = 2;
	b = 7;
	// Copy data to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	// Run kernel - 1 block - 1 thread
	add<<<1,1>>>(d_a, d_b, d_c);
	// Copy data from device
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	// Cleaning
	printf("%d+%d=%d\n",a,b,c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}